#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <direct.h>
#include <math.h>

#include "simpleini\simpleIni.h"
#include "SDL.h"

#undef main

FILE* c_output;
int timeStep;

int N = 100;
int S = 1; // STEPS
bool logStamp;
bool logInit;
bool logChanges;

void loadOptions() {
	// Loads up the config.ini file, and extracts parameters
	printf("Loading Options... \n", N);

	logStamp = true;
	logInit = true;
	logChanges = true;

	CSimpleIniA ini;
	ini.SetUnicode();
	ini.LoadFile("config.ini");
	printf("Completed! \n\nParameters \n", N);
	// Size paramater
	const char * value = ini.GetValue("General", "n", "0");

	// error checking
	if (!atoi(value)) {
		printf("Error Parsing Size, using default value (N = 100) \n");
	}
	else {
		N = atoi(value);
		printf(" N = %d \n", N);
	}

	// Size paramater
	const char * steps = ini.GetValue("General", "s", "1");

	// error checking
	if (!atoi(steps)) {
		printf("Error Parsing Steps, using default value (S = 1) \n");
	}
	else {
		S = atoi(steps);
		printf(" TimeSteps = %d \n", S);
	}

	const char *sizestamp = ini.GetValue("Logs", "sizestamp", "yes");
	if (sizestamp[0] == 'n' || sizestamp[0] == '0' || sizestamp[0] == 'f' || sizestamp[0] == 'F')
		logStamp = false;
	const char *loginit = ini.GetValue("Logs", "loginit", "yes");
	if (loginit[0] == 'n' || loginit[0] == '0' || loginit[0] == 'f' || loginit[0] == 'F')
		logInit = false;
	const char *logchanges = ini.GetValue("Logs", "logchanges", "yes");
	if (logchanges[0] == 'n' || logchanges[0] == '0' || logchanges[0] == 'f' || logchanges[0] == 'F')
		logChanges = false;
}

void pullSpecs(int *blocks, int *threads) {
	int deviceCount, device;
	int gpuDeviceCount = 0;
	struct hipDeviceProp_t properties;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999)
			if (device == 0)
			{
				*blocks = properties.multiProcessorCount;
				*threads = properties.maxThreadsPerMultiProcessor;
				printf("Success!\n\n");
			}
	}
}

__global__ void processChunk(char *value, char* oldFrame, int bm, int br, int tm, int tr, int N, int seed) {
	//int sIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int b = bm;
	if (br > 0 && blockIdx.x < br) {
		b++;
	}

	int t = tm;
	if (tr > 0 && threadIdx.x < tr) {
		t++;
	}

	hiprandState_t state;

	hiprand_init((threadIdx.x + blockIdx.x) * seed / blockIdx.x, 0, 0, &state);

	for (int z = 0; z < b; z++) {
		int y = 0;
		for (int x = 0; x < t; x++) {
			for (int y = 0; y < t; y++) {
				int index = (blockDim.x * x + threadIdx.x % N) + (blockDim.x * y + threadIdx.x / N) * N + (gridDim.x * z + blockIdx.x) * N * N;
				if (index < N*N*N) { 
					// sanity check
					int newValue = hiprand(&state) % 3;
					//Use old values to determine probablility of mutation
					//if (value[index] != newValue) {
						//value[index] = newValue;
						value[index] = 5;
					//}
					//value[index] = blockIdx.x + 5;
				}
			}
		}
	}
}

// EACH CHARACTER POSSIBLY CONTAINS DATA FOR 4 CELLS
// BYTE IS BROKEN AS FOLLOWS :
// 00 / 01 / 10 / 11
int main(int argc, char *args[])
{
	// Load Specs
	int blocks, threads;
	printf("Retreiveing Hardware Specifications... \n");

	pullSpecs(&blocks, &threads);
	printf("multiProcessorCount %d\n", blocks);
	printf("maxThreadsPerMultiProcessor %d\n", threads);
	printf("\n");
	
	// Load Options
	loadOptions();
	printf("\n");

	// Adds numbers and stuff
	int SIZE = N*N*N; // N^3 Size
	char *a;
	char *n_a;
	char *d_a;
	char *d_init;

	a = (char*)malloc(SIZE*sizeof(char));
	n_a = (char*)malloc(SIZE*sizeof(char));

	int check = mkdir("outputs");

	char buff[256];
	if (logStamp)
		sprintf(buff, "outputs/InitState_%d.csv", N);
	else
		sprintf(buff, "outputs/InitState.csv", N);

	FILE* output;
	if (logInit) {
		output = fopen(buff, "w");
		fputs("X,Y,Z,CELLTYPE\n", output);
	}
	srand(time(NULL));

	printf("Generating Initial Population\n");
	//a contains all value
	for (int i = 0; i < SIZE; i++) {
		// if healthy, glia = 10, normal = 00, cancer glia = 11, cancer = 01
		int g = rand() % 2;
		a[i] = (rand() % 2) * (g+g); // everything starts as a healthy cell
		// This is where cell structure will be defined

		if (logInit)
			fprintf(output, "%d,%d,%d,%d\n", i % N, (i / N) % N, i / (N * N), a[i]);
	}
	// write the initial state
	if (logInit)
		fclose(output);
	 
	printf("Success\n\n");

	char c_buff[256];
	if (logStamp)
		sprintf(c_buff, "outputs/ChangeState_%d.csv", N);
	else 
		sprintf(c_buff, "outputs/ChangeState.csv", N);
	if (logChanges) {
		c_output = fopen(c_buff, "w");
		fputs("X,Y,Z,CELLTYPE,TIMESTEP\n", c_output);
	}

	printf("Transferring Data to GPU\n");
	hipMalloc(&d_a, SIZE*sizeof(char));
	hipMalloc(&d_init, SIZE*sizeof(char));
	// have to load initial state, there is no way around it
	hipMemcpy(d_a, a, SIZE*sizeof(char), hipMemcpyHostToDevice); 
	hipMemcpy(d_init, a, SIZE*sizeof(char), hipMemcpyHostToDevice);
	printf("Success\n\n");

	int blockCount = min(N, blocks);
	int threadCount = min(N*N, threads * 13 / 19);

	int bm = N / blockCount;
	int br = N % blockCount;

	int tm = N*N / threadCount;
	int tr = N*N % threadCount;

	printf("Crunching Data\n", tm, tr);
	for (int timeStep = 0; timeStep < S; timeStep++) {
		int seed = rand();
		processChunk <<<blockCount, threadCount>> >(d_a, d_init, bm, br, tm, tr, N, seed);
		hipDeviceSynchronize(); // Force Kernels to complete

		hipMemcpy(n_a, d_a, SIZE*sizeof(char), hipMemcpyDeviceToHost);

		for (int i = 0; i < SIZE; i++) {
			// if healthy, glia = 10, normal = 00, cancer glia = 11, cancer = 01
			if (a[i] != n_a[i])  {
				// only record differences, also update a[i]
				a[i] = n_a[i];
				// Send changes back to the GPU
				if (logChanges)
					fprintf(c_output, "%d,%d,%d,%d,%d\n", i % N, (i / N) % N, i / (N * N), n_a[i], timeStep);
				hipMemcpy(&d_init[i], &a[i], sizeof(char), hipMemcpyHostToDevice);
			}
		}
	}

	if (logChanges)
		fclose(c_output);
	
	printf("Simulation Completed\n");

	while (true) {

	}
	/*
	int screenWidth = 640;
	int screenHeight = 480;

	SDL_Window* pWindow = NULL;
	pWindow = SDL_CreateWindow("Brain Cancer Start", SDL_WINDOWPOS_UNDEFINED,
		SDL_WINDOWPOS_UNDEFINED,
		screenWidth,
		screenHeight,
		SDL_WINDOW_SHOWN);

	SDL_Renderer* pRender = SDL_CreateRenderer(pWindow, -1, 0);

	SDL_Event evt;
	bool alive = true;
	while (alive) {
		while (SDL_PollEvent(&evt)) {
			if (evt.type == SDL_QUIT) {
				alive = false;
				break;
			}
			if (evt.type == SDL_KEYDOWN && evt.key.keysym.sym == SDLK_ESCAPE) {
				alive = false;
				break;
			}
		}
		int square = 40;
		int width = (screenWidth / square);
		int height = (screenHeight / square);

		hipMemcpy(d_a, a, SIZE*sizeof(char), hipMemcpyHostToDevice);

		//addChange << <1, SIZE >> >(d_a, SIZE);

		hipMemcpy(a, d_a, SIZE*sizeof(char), hipMemcpyDeviceToHost);

		for (int core = 0; core < SIZE; core++) {
			int i = core % width;
			
			int j = 0;
			if (core >= width) {
				j = core / width;
			}
			for (int x = 0; x < square; x++) {
				for (int y = 0; y < square; y++) {
					SDL_SetRenderDrawColor(pRender, a[core], 105, 180, 255);
					SDL_RenderDrawPoint(pRender, i * square + x, j * square + y);
				}
			}

		}
		SDL_RenderPresent(pRender);
	}

	SDL_DestroyWindow(pWindow);
	SDL_Quit();
	*/
	free(a);
	hipFree(d_a);
	return 0;
}

